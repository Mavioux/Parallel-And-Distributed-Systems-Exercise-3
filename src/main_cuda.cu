#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define CHANNEL_NUM 1

#define MAX_IMAGE_VALUE 255
#define MIN_IMAGE_VALUE 0
#define PI 3.1415926536

double AWGN_generator2()
{ /* Generates additive white Gaussian Noise samples with zero mean and a standard deviation of 1. */

    double temp1;
    double temp2;
    double result;
    int p;
    double noise_sigma = 0.02;

    p = 1;

    while (p > 0)
    {
        temp2 = (rand() / ((double)RAND_MAX)); /*  rand() function generates an
                                                       integer between 0 and  RAND_MAX,
                                                       which is defined in stdlib.h.
                                                   */

        if (temp2 == 0)
        { // temp2 is >= (RAND_MAX / 2)
            p = 1;
        } // end if
        else
        { // temp2 is < (RAND_MAX / 2)
            p = -1;
        } // end else

    } // end while()

    temp1 = cos((2.0 * (double)PI) * rand() / ((double)RAND_MAX));
    result = sqrt(-2.0 * log(temp2)) * temp1;

    return result * noise_sigma; // return the generated random sample to the caller
}

double **non_local_means(double **input_image, int patchsize, double filter_sigma, double patch_sigma, int width, int height)
{
    double **output_image = (double **)malloc(height * sizeof(double *));
    for (int i = 0; i < height; i++)
    {
        output_image[i] = (double *)malloc(width * sizeof(double));
    }

    /* Loop for each pixel that is inside the patchsize limits */
    for (int i = patchsize / 2; i < height - patchsize / 2; i++)
    {
        for (int j = patchsize / 2; j < width - patchsize / 2; j++)
        {
            /* Create the patchsize * patchsize grid with the selected pixel at the centre */
            double **pixel_patch = (double **)malloc(patchsize * sizeof(double *));
            for (int i = 0; i < patchsize; i++)
            {
                pixel_patch[i] = (double *)malloc(patchsize * sizeof(double));
            }
            int counter_i = 0;
            for (int k = -patchsize / 2; k < patchsize / 2 + 1; k++)
            {
                int counter_j = 0;
                for (int l = -patchsize / 2; l < patchsize / 2 + 1; l++)
                {
                    pixel_patch[counter_i][counter_j++] = input_image[i + k][j + l];
                }
                counter_i++;
            }

            /* Initialize the ouput image value to zero */
            output_image[i][j] = 0;
            double zeta = 0;
            double zeta2 = 0;
            /* Comparison patch (we take into account ourselves too) */
            for (int m = patchsize / 2; m < height - patchsize / 2; m++)
            {
                for (int n = patchsize / 2; n < width - patchsize / 2; n++)
                {
                    /* Create the patchsize * patchsize grid with the selected pixel at the centre */
                    double **comparison_patch = (double **)malloc(patchsize * sizeof(double *));
                    for (int i = 0; i < patchsize; i++)
                    {
                        comparison_patch[i] = (double *)malloc(patchsize * sizeof(double));
                    }
                    int counter_i = 0;
                    for (int k = -patchsize / 2; k < patchsize / 2 + 1; k++)
                    {
                        int counter_j = 0;
                        for (int l = -patchsize / 2; l < patchsize / 2 + 1; l++)
                        {
                            comparison_patch[counter_i][counter_j++] = input_image[m + k][n + l];
                        }
                        counter_i++;
                    }

                    /* Here we should implement the f algorithm */
                    double difference_squared = 0;
                    for (int a = 0; a < patchsize; a++)
                    {
                        for (int b = 0; b < patchsize; b++)
                        {
                            double distX = (a - patchsize / 2) * (a - patchsize / 2);
                            double distY = (b - patchsize / 2) * (b - patchsize / 2);
                            double dist = -(distX + distY) / (patch_sigma * patch_sigma);
                            dist = exp(dist);

                            difference_squared += dist * (pixel_patch[a][b] - comparison_patch[a][b]) * (pixel_patch[a][b] - comparison_patch[a][b]);
                        }
                    }

                    double w_difference_squared = -difference_squared / (filter_sigma * filter_sigma);

                    double w = exp(w_difference_squared);
                    zeta += w;

                    output_image[i][j] += input_image[m][n] * w;
                }
            }
            output_image[i][j] = output_image[i][j] / zeta;
        }
    }

    return output_image;
}

int main()
{
    int width, height, bpp;
    int patchsize = 3;
    time_t t;

    /* Intializes random number generator */
    srand((unsigned)time(&t));

    uint8_t *original_image = stbi_load("../images/musk.jpg", &width, &height, &bpp, 1);
    double *normalized_image = (double *)malloc(width * height * sizeof(double));
    double *noisy_image = (double *)malloc(width * height * sizeof(double));
    uint8_t *noisy_image_for_save = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    double **normalized_noisy_2D = (double **)malloc(height * sizeof(double *));
    for (int i = 0; i < height; i++)
    {
        normalized_noisy_2D[i] = (double *)malloc(width * sizeof(double));
    }

    double **normalized_denoised_2D = (double **)malloc(height * sizeof(double *));
    for (int i = 0; i < height; i++)
    {
        normalized_denoised_2D[i] = (double *)malloc(width * sizeof(double));
    }

    uint8_t *denoised_image = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    /* Image Normalization and Noise Addition*/
    for (int i = 0; i < width * height; i++)
    {
        normalized_image[i] = (double)original_image[i] / (double)(MAX_IMAGE_VALUE - MIN_IMAGE_VALUE + 1);
        noisy_image[i] = normalized_image[i] + AWGN_generator2();
        //noisy_image[i] = normalized_image[i];
        if (noisy_image[i] > 1)
        {
            noisy_image[i] = 1;
        }

        if (noisy_image[i] < 0)
        {
            noisy_image[i] = 0;
        }
    }

    /* Denormalize and Save the noisy image */
    for (int i = 0; i < width * height; i++)
    {
        noisy_image_for_save[i] = (uint8_t)(noisy_image[i] * 255);
    }

    stbi_write_jpg("../images/fight_black_white.jpg", width, height, 1, original_image, 0);
    stbi_write_jpg("../images/noisy_image.jpg", width, height, 1, noisy_image_for_save, 0);

    /* Map to 2D */
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            normalized_noisy_2D[i][j] = noisy_image[i * width + j];
        }
    }

    normalized_denoised_2D = non_local_means(normalized_noisy_2D, patchsize, 0.2, 1.67, width, height);

    /* Denormalize and map into 1D the denoised image */
    int counter = 0;
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            denoised_image[counter++] = normalized_denoised_2D[i][j] * 255;
            // denoised_image[counter++] = normalized_denoised_2D[i][j] * 0;
        }
    }

    stbi_write_jpg("../images/denoised_image.jpg", width, height, CHANNEL_NUM, denoised_image, 0);

    stbi_image_free(original_image);
    free(noisy_image_for_save);
    free(normalized_image);
    free(noisy_image);

    return 0;
}