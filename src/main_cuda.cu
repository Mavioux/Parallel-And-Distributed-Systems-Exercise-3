#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define CHANNEL_NUM 1

#define MAX_IMAGE_VALUE 255
#define MIN_IMAGE_VALUE 0
#define PI 3.1415926536

float AWGN_generator2()
{ /* Generates additive white Gaussian Noise samples with zero mean and a standard deviation of 1. */

    float temp1;
    float temp2;
    float result;
    int p;
    float noise_sigma = 0.02;

    p = 1;

    while (p > 0)
    {
        temp2 = (rand() / ((float)RAND_MAX)); /*  rand() function generates an
                                                       integer between 0 and  RAND_MAX,
                                                       which is defined in stdlib.h.
                                                   */

        if (temp2 == 0)
        { // temp2 is >= (RAND_MAX / 2)
            p = 1;
        } // end if
        else
        { // temp2 is < (RAND_MAX / 2)
            p = -1;
        } // end else

    } // end while()

    temp1 = cos((2.0 * (float)PI) * rand() / ((float)RAND_MAX));
    result = sqrt(-2.0 * log(temp2)) * temp1;

    return result * noise_sigma; // return the generated random sample to the caller
}

__global__ void pixel_algorithm(float *input_image_gpu, float *output_image_gpu, float *pixel_patch_gpu, float *comparison_patch_gpu, int width, int height, int patchsize, float filter_sigma, float patch_sigma) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int i = ix / height;
    int j = ix % width;

    // printf("width - patchsize / 2 - 1: %d\n", width - patchsize / 2 - 1);

    

    // Return if out of computation bounds
    if(i < patchsize / 2 || i > width - patchsize / 2 - 1) {
        // printf("Returning for ix: %d\t i: %d\t j: %d\n", ix, i , j);
        return;
    }
    if(j < patchsize / 2 || j > height - patchsize / 2 - 1) {
        // printf("Returning for ix: %d\t i: %d\t j: %d\n", ix, i , j);
        return;
    }
    // printf("%d\t i: %d\t j: %d\n", ix, i , j);

    /* Create the patchsize * patchsize grid with the selected pixel at the centre */
    int counter_i = 0;
    for (int k = -patchsize / 2; k < patchsize / 2 + 1; k++)
    {
        int counter_j = 0;
        for (int l = -patchsize / 2; l < patchsize / 2 + 1; l++)
        {
            pixel_patch_gpu[counter_i * (patchsize) + counter_j] = input_image_gpu[(i + k) * width + (j + l)];
            counter_j++;
        }
        counter_i++;
    }

    /* Initialize the ouput image value to zero */
    output_image_gpu[i * width + j] = 0;
    float zeta = 0;

    for (int m = patchsize / 2; m < height - patchsize / 2; m++)
    {
        for (int n = patchsize / 2; n < width - patchsize / 2; n++)
        {
            /* Create the patchsize * patchsize grid with the selected pixel at the centre */
            int counter_i = 0;
            for (int k = -patchsize / 2; k < patchsize / 2 + 1; k++)
            {
                int counter_j = 0;
                for (int l = -patchsize / 2; l < patchsize / 2 + 1; l++)
                {
                    comparison_patch_gpu[counter_i * (patchsize) + counter_j] = input_image_gpu[(m + k) * width + (n + l)];
                    counter_j++;
                }
                counter_i++;
            }

            /* Here we should implement the f algorithm */
            float difference_squared = 0;
            for (int a = 0; a < patchsize; a++)
            {
                for (int b = 0; b < patchsize; b++)
                {
                    float distX = (a - patchsize / 2) * (a - patchsize / 2);
                    float distY = (b - patchsize / 2) * (b - patchsize / 2);
                    float dist = -(distX + distY) / (patch_sigma * patch_sigma);
                    dist = exp(dist);
                    //used to be patchsize / 2 but could not remember why so i reverted it
                    difference_squared += dist * (pixel_patch_gpu[a * (patchsize) + b] - comparison_patch_gpu[a * (patchsize) + b]) * (pixel_patch_gpu[a * (patchsize) + b] - comparison_patch_gpu[a * (patchsize) + b]);
                }
            }

            float w_difference_squared = -difference_squared / (filter_sigma * filter_sigma);

            float w = exp(w_difference_squared);
            zeta += w;

            output_image_gpu[i * width + j] += input_image_gpu[m * width + n] * w;
        }
    }
    output_image_gpu[i * width + j] = output_image_gpu[i * width + j] / zeta;
    printf("%d\t i: %d\t j: %d\t output_image_gpu[%d]: %f\n", ix, i , j, i * width + j, output_image_gpu[i * width + j]);
}

float *non_local_means(float *input_image, int patchsize, float filter_sigma, float patch_sigma, int width, int height)
{
    float *output_image = (float *)malloc(height * width * sizeof(float));

    float *output_image_gpu;
    hipMalloc(&output_image_gpu, height * width * sizeof(float));

    float *input_image_gpu;
    hipMalloc(&input_image_gpu, height * width * sizeof(float));

    float *pixel_patch_gpu;
    hipMalloc(&pixel_patch_gpu, patchsize * patchsize * sizeof(float));

    float *comparison_patch_gpu;
    hipMalloc(&comparison_patch_gpu, patchsize * patchsize * sizeof(float));

    // hipMemcpy(output_image_gpu, output_image, height * width * sizeof(float), hipMemcpyHostToDevice); // This might be useless in the beggining
    hipMemcpy(input_image_gpu, input_image, height * width * sizeof(float), hipMemcpyHostToDevice);

    // Technically here we would like to initiate width * height number of threads in order to compute in parallel the output value of each i in width*size
    pixel_algorithm<<<width, height>>>(input_image_gpu, output_image_gpu, pixel_patch_gpu, comparison_patch_gpu, width, height, patchsize, filter_sigma , patch_sigma);

    hipMemcpy(output_image, output_image_gpu, height * width * sizeof(float), hipMemcpyDeviceToHost);
    return output_image;
}

int main()
{
    int width, height, bpp;
    int patchsize = 3;
    time_t t;

    /* Intializes random number generator */
    srand((unsigned)time(&t));

    uint8_t *original_image = stbi_load("../images/musk.jpg", &width, &height, &bpp, 1);
    float *normalized_image = (float *)malloc(width * height * sizeof(float));
    float *noisy_image = (float *)malloc(width * height * sizeof(float));
    float *denoised_image_float = (float *)malloc(width * height * sizeof(float));
    uint8_t *noisy_image_for_save = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    // float **normalized_noisy_2D = (float **)malloc(height * sizeof(float *));
    // for (int i = 0; i < height; i++)
    // {
    //     normalized_noisy_2D[i] = (float *)malloc(width * sizeof(float));
    // }

    // float **normalized_denoised_2D = (float **)malloc(height * sizeof(float *));
    // for (int i = 0; i < height; i++)
    // {
    //     normalized_denoised_2D[i] = (float *)malloc(width * sizeof(float));
    // }

    uint8_t *denoised_image = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    /* Image Normalization and Noise Addition*/
    for (int i = 0; i < width * height; i++)
    {
        normalized_image[i] = (float)original_image[i] / (float)(MAX_IMAGE_VALUE - MIN_IMAGE_VALUE + 1);
        noisy_image[i] = normalized_image[i] + AWGN_generator2();
        //noisy_image[i] = normalized_image[i];
        if (noisy_image[i] > 1)
        {
            noisy_image[i] = 1;
        }

        if (noisy_image[i] < 0)
        {
            noisy_image[i] = 0;
        }
    }

    /* Denormalize and Save the noisy image */
    for (int i = 0; i < width * height; i++)
    {
        noisy_image_for_save[i] = (uint8_t)(noisy_image[i] * 255);
    }

    stbi_write_jpg("../images/fight_black_white.jpg", width, height, 1, original_image, 0);
    stbi_write_jpg("../images/noisy_image.jpg", width, height, 1, noisy_image_for_save, 0);

    // /* Map to 2D */
    // for (int i = 0; i < height; i++)
    // {
    //     for (int j = 0; j < width; j++)
    //     {
    //         normalized_noisy_2D[i][j] = noisy_image[i * width + j];
    //     }
    // }

    denoised_image_float = non_local_means(noisy_image, patchsize, 0.2, 1.67, width, height);

    /* Denormalize and Save the denoised image */
    for (int i = 0; i < width * height; i++)
    {
        denoised_image[i] = (uint8_t)(denoised_image_float[i] * 255);
    }

    // /* Denormalize and map into 1D the denoised image */
    // int counter = 0;
    // for (int i = 0; i < height; i++)
    // {
    //     for (int j = 0; j < width; j++)
    //     {
    //         denoised_image[counter++] = normalized_denoised_2D[i][j] * 255;
    //         // denoised_image[counter++] = normalized_denoised_2D[i][j] * 0;
    //     }
    // }

    stbi_write_jpg("../images/denoised_image.jpg", width, height, CHANNEL_NUM, denoised_image, 0);
    
    printf("Width: %d\nHeight: %d\nTotal Kernels", width, height, width * height);

    stbi_image_free(original_image);
    free(noisy_image_for_save);
    free(normalized_image);
    free(noisy_image);

    return 0;
}