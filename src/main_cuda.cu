#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define CHANNEL_NUM 1

#define MAX_IMAGE_VALUE 255
#define MIN_IMAGE_VALUE 0
#define PI 3.1415926536

/**
 * Function that generates additive white Gaussian Noise samples
 * with zero mean and a standard deviation of 1. 
 * Input: 
 *      No input values
 * Output:
 *      float result * noise_sigma : The generated random sample to the caller
 *       
*/
float AWGN_generator()
{
    float temp1;
    float temp2;
    float result;
    int p;
    float noise_sigma = 0.02;

    p = 1;

    while (p > 0)
    {
        temp2 = (rand() / ((float)RAND_MAX)); /*  rand() function generates an
                                                  integer between 0 and  RAND_MAX,
                                                  which is defined in stdlib.h.
                                              */

        if (temp2 == 0)
        { // temp2 is >= (RAND_MAX / 2)
            p = 1;
        } // end if
        else
        { // temp2 is < (RAND_MAX / 2)
            p = -1;
        } // end else

    } // end while()

    temp1 = cos((2.0 * (float)PI) * rand() / ((float)RAND_MAX));
    result = sqrt(-2.0 * log(temp2)) * temp1;

    return result * noise_sigma;
}

/** 
* Function that implements the Non Local Means parallel algorithm for every pixel.
* At first we create the patchsize * patchsize grid with the selected pixel at the centre.
* Then we run a nested for loop in order to get the pixel patch and compare it our pixel patch.
* After the calculation we save the results in the output image and we return it.
* Input:
*       float *input_image_gpu      : values from every pixel for the input image
*       float *output_image_gpu     : values from every pixel for the output image
*       float *pixel_patch_gpu      : values from every pixel for the patch 
*       float *comparison_patch_gpu : values from every pixel for the comparison patch 
*       int width                   : value for the image width
*       int height                  : value for the image height
* Output:
*       No output values
**/
__global__ void pixel_algorithm(float *input_image_gpu, float *output_image_gpu, float *pixel_patch_gpu, float *comparison_patch_gpu, int width, int height, int patchsize, float filter_sigma, float patch_sigma) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int i = ix / height;
    int j = ix % width;


    /* Return if out of computation bounds */
    if(i < patchsize / 2 || i > width - patchsize / 2 - 1) {
        return;
    }
    if(j < patchsize / 2 || j > height - patchsize / 2 - 1) {
        return;
    }

    /* Create the patchsize * patchsize grid with the selected pixel at the centre */
    int counter_i = 0;
    for (int k = -patchsize / 2; k < patchsize / 2 + 1; k++)
    {
        int counter_j = 0;
        for (int l = -patchsize / 2; l < patchsize / 2 + 1; l++)
        {
            pixel_patch_gpu[ix * (patchsize * patchsize) + counter_i * (patchsize) + counter_j] = input_image_gpu[(i + k) * width + (j + l)];
            counter_j++;
        }
        counter_i++;
    }

    /* Initialize the ouput image value to zero */
    output_image_gpu[i * width + j] = 0;
    float zeta = 0;

    for (int m = patchsize / 2; m < height - patchsize / 2; m++)
    {
        for (int n = patchsize / 2; n < width - patchsize / 2; n++)
        {
            /* Create the patchsize * patchsize grid with the selected pixel at the centre */
            int counter_i = 0;
            for (int k = -patchsize / 2; k < patchsize / 2 + 1; k++)
            {
                int counter_j = 0;
                for (int l = -patchsize / 2; l < patchsize / 2 + 1; l++)
                {
                    comparison_patch_gpu[ix * (patchsize * patchsize) + counter_i * (patchsize) + counter_j] = input_image_gpu[(m + k) * width + (n + l)];
                    counter_j++;
                }
                counter_i++;
            }

            /* Here we implement the Non Local Means algorithm */
            float difference_squared = 0;
            for (int a = 0; a < patchsize; a++)
            {
                for (int b = 0; b < patchsize; b++)
                {
                    float distX = (a - patchsize / 2) * (a - patchsize / 2);
                    float distY = (b - patchsize / 2) * (b - patchsize / 2);
                    float dist = -(distX + distY) / (patch_sigma * patch_sigma);
                    dist = exp(dist);
                    // used to be patchsize / 2 but could not remember why so i reverted it
                    difference_squared += dist * (pixel_patch_gpu[ix * (patchsize * patchsize) + a * (patchsize) + b] - comparison_patch_gpu[ix * (patchsize * patchsize) + a * (patchsize) + b]) * (pixel_patch_gpu[ix * (patchsize * patchsize) + a * (patchsize) + b] - comparison_patch_gpu[ix * (patchsize * patchsize) + a * (patchsize) + b]);
                }
            }

            float w_difference_squared = -difference_squared / (filter_sigma * filter_sigma);

            float w = exp(w_difference_squared);
            zeta += w;

            output_image_gpu[i * width + j] += input_image_gpu[m * width + n] * w;
        }
    }
    /* Save the output value after the calculation */
    output_image_gpu[i * width + j] = output_image_gpu[i * width + j] / zeta;
}

/** 
* Function that implements the Non Local Means parallel algorithm for the image.
* At first we allocate memory for the output image, for the pixel patch and for the comparison patch.
* Then we run the pixel_algorithm for every pixel in order to calculate all the needed values.
* Input:
*       float **input_image : values from every pixel for the input image
*       int patchsize       : value from the patch size
*       float filter_sigma  : value from the filter sigma
*       float patch_sigma   : value from the patch sigma
*       int width           : value for the image width
*       int height          : value for the image height
* Output:
*       float *output_image : values from every pixel for the output image
**/
float *non_local_means(float *input_image, int patchsize, float filter_sigma, float patch_sigma, int width, int height)
{
    float *output_image = (float *)malloc(height * width * sizeof(float));

    float *output_image_gpu;
    hipMalloc(&output_image_gpu, height * width * sizeof(float));

    float *input_image_gpu;
    hipMalloc(&input_image_gpu, height * width * sizeof(float));

    float *pixel_patch_gpu;
    hipMalloc(&pixel_patch_gpu, width * height * patchsize * patchsize * sizeof(float));

    float *comparison_patch_gpu;
    hipMalloc(&comparison_patch_gpu, width * height * patchsize * patchsize * sizeof(float));

    hipMemcpy(input_image_gpu, input_image, height * width * sizeof(float), hipMemcpyHostToDevice);

    // Technically here we would like to initiate width * height number of threads in order to compute in parallel the output value of each i in width*size
    pixel_algorithm<<<width, height>>>(input_image_gpu, output_image_gpu, pixel_patch_gpu, comparison_patch_gpu, width, height, patchsize, filter_sigma , patch_sigma);

    hipMemcpy(output_image, output_image_gpu, height * width * sizeof(float), hipMemcpyDeviceToHost);
    return output_image;
}

/**
 * Main has 4 command line arguments
 * Argument 1: Path for jpg file, Argument 2: Patch size value, Argument 3: Filter Sigma value, Argument 4: Patch Sigma value
 * 
*/
int main(int argc, char **argv)
{   
    /* Initialize the variables by using the arguments */
    int width, height, bpp;
    int patchsize = atoi(argv[2]);

    time_t t;

    char *image_file_name = argv[1];
    char buf[512];

    float filter_signa = atof(argv[3]); // 0.2
    float patch_sigma = atof(argv[4]);  // 1.67

    /* Initialize random number generator */
    srand((unsigned)time(&t));

     /* Load the original image and allocate memory for the normalized, the noisy and the denoised images */
    uint8_t *original_image = stbi_load(image_file_name, &width, &height, &bpp, 1);
    float *normalized_image = (float *)malloc(width * height * sizeof(float));
    float *noisy_image = (float *)malloc(width * height * sizeof(float));
    float *denoised_image_float = (float *)malloc(width * height * sizeof(float));
    uint8_t *noisy_image_for_save = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    uint8_t *denoised_image = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    /* Image Normalization and Noise Addition*/
    for (int i = 0; i < width * height; i++)
    {
        normalized_image[i] = (float)original_image[i] / (float)(MAX_IMAGE_VALUE - MIN_IMAGE_VALUE + 1);
        noisy_image[i] = normalized_image[i] + AWGN_generator();
        
        /* Check and avoid getting out of the limit colors */
        if (noisy_image[i] > 1)
        {
            noisy_image[i] = 1;
        }

        if (noisy_image[i] < 0)
        {
            noisy_image[i] = 0;
        }
    }

    /* Denormalize and Save the noisy image */
    for (int i = 0; i < width * height; i++)
    {
        noisy_image_for_save[i] = (uint8_t)(noisy_image[i] * 255);
    }

    snprintf(buf, sizeof buf, "%s%s_%s", "../images/", image_file_name, "black_white.jpg");
    stbi_write_jpg(buf, width, height, 1, original_image, 0);
    snprintf(buf, sizeof buf, "%s%s_%s", "../images/", image_file_name, "noisy.jpg");
    stbi_write_jpg(buf, width, height, 1, noisy_image_for_save, 0);

    /* Start measuring time */
    clock_t begin = clock();

    /* Run algorith that calculates the denoised image */
    denoised_image_float = non_local_means(noisy_image, patchsize, 0.2, 1.67, width, height);

    /* Stop measuring time */  
    clock_t end = clock();
    double duration = (double)(end - begin) / CLOCKS_PER_SEC;

    printf("Duration: %f\n", duration);

    /* Denormalize and Save the denoised image */
    for (int i = 0; i < width * height; i++)
    {
        denoised_image[i] = (uint8_t)(denoised_image_float[i] * 255);
    }

    /* Save the noise denoised image */
    snprintf(buf, sizeof buf, "%s%s_%s", "../images/", image_file_name, "denoised.jpg");
    stbi_write_jpg(buf, width, height, CHANNEL_NUM, denoised_image, 0);

    /* Deallocate used memory */
    stbi_image_free(original_image);
    free(noisy_image_for_save);
    free(normalized_image);
    free(noisy_image);

    return 0;
}
